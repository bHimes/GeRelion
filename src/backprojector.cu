#include "hip/hip_runtime.h"
/***************************************************************************
 *
 * Author : "Huayou SU, Wen WEN, Xiaoli DU, Dongsheng LI"
 * Parallel and Distributed Processing Laboratory of NUDT
 * Author : "Maofu LIAO"
 * Department of Cell Biology, Harvard Medical School
 *
 * This file is the GPU program for backproject, 
 * including the kernels and host side program.
 * We implemented the key function  reconstruct with GPU, named reconstruct_gpu.
 * Some of the data structure and aux functions are from Relion
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * This complete copyright notice must be included in any revised version of the
 * source code. Additional authorship citations may be added, but existing
 * author citations must be preserved.
 ***************************************************************************/


#include "src/backprojector.h"
#include "src/math_function.h"

__constant__ DOUBLE __L_array [100 * 4 * 4];
__constant__ DOUBLE __R_array [100 * 4 * 4];

#ifndef FLOAT_PRECISION
__device__ double atomicAdd_double(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do
	{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	}
	while (assumed != old);

	return __longlong_as_double(old);
}
#endif
void compare_CPU_GPU(Complex* in_cpu, CUFFT_COMPLEX * in_gpu, int size_)
{
	Complex* in_gpu_H;
	in_gpu_H = (Complex*) malloc(size_ * sizeof(Complex));
	hipMemcpy(in_gpu_H, in_gpu, size_ * sizeof(Complex), hipMemcpyDeviceToHost);

	for (int i = 0; i < size_; i++)
	{
		if (abs(in_cpu[i] - in_gpu_H[i]) > 0.00000001)
		{
			std::cout << "Resuls Error at real : " << i << "  " << in_cpu[i].real << "  " << in_gpu_H[i].real << std::endl;
			std::cout << "Resuls Error at imag: " << i << "  " << in_cpu[i].imag << "  " << in_gpu_H[i].imag << std::endl;
			for (int j = i; j < i + 40; j++)
			{
				std::cout << "Resuls Error at real : " << j << "  " << in_cpu[j].real << "  " << in_gpu_H[j].real << std::endl;
				std::cout << "Resuls Error at imag: " << j << "  " << in_cpu[j].imag << "  " << in_gpu_H[j].imag << std::endl;
			}

			REPORT_ERROR("ERROR: in_cpu[i]!=in_gpu_H[i] ");

		}

	}
	free(in_gpu_H);
}

void compare_CPU_GPU(DOUBLE* in_cpu, DOUBLE* in_gpu, int size_)
{
	DOUBLE* in_gpu_H;
	in_gpu_H = (DOUBLE*) malloc(size_ * sizeof(DOUBLE));
	hipMemcpy(in_gpu_H, in_gpu, size_ * sizeof(DOUBLE), hipMemcpyDeviceToHost);

	for (int i = 0; i < size_; i++)
	{
		if (abs(in_cpu[i] - in_gpu_H[i]) > 0.00000001)
		{
			std::cout << "Resuls Error at: " << i << "  " << in_cpu[i] << "  " << in_gpu_H[i] << std::endl;
			REPORT_ERROR("ERROR: in_cpu[i]!=in_gpu_H[i] ");
		}
	}
	free(in_gpu_H);
}

__global__ void update_tau2_with_fsc_kernel(const  DOUBLE* __restrict__ sigma2_D, DOUBLE* fsc_D, DOUBLE* tau2_D, DOUBLE* data_vs_prior_D, int data_size, bool is_whole_instead_of_half)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (global_index >= data_size)
	{
		return;
	}
	DOUBLE myfsc = (0.001 >= fsc_D[global_index]) ? (0.001) : fsc_D[global_index];
	if (is_whole_instead_of_half)
	{
		// Factor two because of twice as many particles
		// Sqrt-term to get 60-degree phase errors....
		myfsc = sqrt(2. * myfsc / (myfsc + 1.));
	}
	myfsc = (myfsc >= 0.999) ? (0.999) : myfsc;
	DOUBLE myssnr = myfsc / (1. - myfsc);
	DOUBLE fsc_based_tau = myssnr * sigma2_D[global_index];
	tau2_D[global_index] = fsc_based_tau;
	// data_vs_prior is merely for reporting: it is not used for anything in the reconstruction
	data_vs_prior_D[global_index] = myssnr;
}
void update_tau2_with_fsc_gpu(DOUBLE* sigma2_D, DOUBLE* fsc_D, DOUBLE* tau2_D, DOUBLE* data_vs_prior_D, int data_size, bool is_whole_instead_of_half)
{
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((data_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);

	update_tau2_with_fsc_kernel <<< gridDim, blockDim>>>(sigma2_D, fsc_D, tau2_D, data_vs_prior_D, data_size, is_whole_instead_of_half);

}

__global__ void Applymap_additional_to_weight_kernel(DOUBLE* weight_D, DOUBLE* tau2_D, DOUBLE* data_vs_prior_D, DOUBLE* counter_D,
                                                     int max_r2, int xdim, int ydim, int zdim,
                                                     int padding_factor, DOUBLE oversampling_correction, DOUBLE tau2_fudge, bool update_tau2_with_fsc, int minres_map)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	int i, j, k;
	int ip, jp, kp;
	j = global_index % xdim ;
	i = (global_index / xdim) % ydim;
	k =  global_index / (xdim * ydim);

	jp = j;
	ip = (i < xdim) ? i : (i - ydim);
	kp = (k < xdim) ? k : (k - zdim);
	int r2 = (kp * kp + ip * ip + jp * jp);
	if (global_index >= (xdim * ydim * zdim) || r2 >= max_r2)
	{
		return;
	}
	DOUBLE invtau2;
	int ires = ((sqrt((DOUBLE)r2) / padding_factor) > 0) ? (int)((sqrt((DOUBLE)r2) / padding_factor) + 0.5) : (int)((sqrt((DOUBLE)r2) / padding_factor) - 0.5);

	DOUBLE invw = weight_D[global_index];
	// We consider that the values of tau2 will not be negative
	invtau2 = (tau2_D[ires] > 0.) ? (1. / (oversampling_correction * tau2_fudge * tau2_D[ires])) : (1. / (0.001 * invw));

	if (!update_tau2_with_fsc)
	{
#ifdef FLOAT_PRECISION
		atomicAdd(&(data_vs_prior_D[ires]), (DOUBLE) invw / invtau2);
#else
		atomicAdd_double(&(data_vs_prior_D[ires]), (DOUBLE) invw / invtau2);
#endif
	}
#ifdef FLOAT_PRECISION
		atomicAdd(&(counter_D[ires]), (DOUBLE) 1.0);
#else
		atomicAdd_double(&(counter_D[ires]), (DOUBLE) 1.0);
#endif
	if (ires >= minres_map)
	{
		weight_D[global_index] = invw + invtau2;
	}
}
void Applymap_additional_to_weight_gpu(DOUBLE* weight_D, DOUBLE* tau2_D, DOUBLE* data_vs_prior_D, DOUBLE* counter_D,
                                       int max_r2, int xdim, int ydim, int zdim,
                                       int padding_factor, DOUBLE oversampling_correction, DOUBLE tau2_fudge, bool update_tau2_with_fsc, int minres_map)

{
	int model_size = xdim * ydim * zdim;
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((model_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	Applymap_additional_to_weight_kernel <<< gridDim, blockDim>>>(weight_D, tau2_D, data_vs_prior_D, counter_D,
	                                                              max_r2,  xdim,  ydim, zdim,
	                                                              padding_factor,  oversampling_correction,  tau2_fudge,  update_tau2_with_fsc,  minres_map);


}

__global__ void Average_data_vs_prior_kernel(DOUBLE* data_vs_prior_D, const DOUBLE* __restrict__ counter_D, int data_size, int r_max)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= data_size)
	{
		return;
	}

	if (i > r_max)
	{
		data_vs_prior_D[i] = 0.;
	}
	else if (counter_D[i] < 0.001)
	{
		data_vs_prior_D[i] = 999.;
	}
	else
	{
		data_vs_prior_D[i] /= counter_D[i];
	}
}
void Average_data_vs_prior_gpu(DOUBLE* data_vs_prior_D, DOUBLE* counter_D, int data_size, int r_max)
{
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((data_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	Average_data_vs_prior_kernel <<< gridDim,  blockDim>>>(data_vs_prior_D, counter_D, data_size, r_max) ;
}

__global__ void do_normalise_data_kernel(CUFFT_COMPLEX * data_D, int data_size, DOUBLE normalise_value)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i > data_size)
	{
		return;
	}
	data_D[i].x = data_D[i].x / normalise_value;
	data_D[i].y = data_D[i].y / normalise_value;
}
__global__ void do_normalise_weight_kernel(DOUBLE* weight_D, int weight_size, DOUBLE normalise_value)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i > weight_size)
	{
		return;
	}
	weight_D[i] = weight_D[i] / normalise_value;
}
void do_normalise_weight_data_gpu(DOUBLE* weight_D, CUFFT_COMPLEX * data_D, int weight_size, int data_size, DOUBLE normalise_value)
{
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((data_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	do_normalise_data_kernel <<< gridDim , blockDim>>>(data_D, data_size, normalise_value);

	dim3 gridDim2((weight_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	do_normalise_weight_kernel <<< gridDim2 , blockDim>>>(weight_D, weight_size, normalise_value);
}

__global__ void init_Fnewweight_kernel(double* Fnewweight_D, int xdim, int ydim, int zdim, int my_rmax2)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if (global_index > xdim * ydim * zdim)
	{
		return;
	}

	int i, j, k;
	int ip, jp, kp;
	j = global_index % xdim ;
	i = (global_index / xdim) % ydim;
	k = global_index / (xdim * ydim);

	jp = j;
	ip = (i < xdim) ? i : (i - ydim);
	kp = (k < xdim) ? k : (k - zdim);
	int r2 = kp * kp + ip * ip + jp * jp;
	if (r2 < my_rmax2)
	{
		Fnewweight_D[global_index] = 1.0;
	}
	else
	{
		Fnewweight_D[global_index] = 0.0;
	}
}
void init_Fnewweight_gpu(double* Fnewweight_D, int xdim, int ydim, int zdim, int my_rmax2)
{
	int data_size = xdim * ydim * zdim;
	dim3  blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((data_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);

	init_Fnewweight_kernel <<< gridDim, blockDim>>>(Fnewweight_D, xdim, ydim, zdim, my_rmax2);
}
__global__ void calculate_sigma2_kernel(const DOUBLE* __restrict__ weight_D, DOUBLE* sigma2_D, DOUBLE* counter_D, int padding_factor, DOUBLE oversampling_correction,
                                        int xdim, int ydim, int zdim, int max_r2, int size_sigma)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	int i, j, k; // x, y, z;
	int ip, jp, kp;
	j = global_index % xdim;
	i = (global_index / xdim) % ydim;
	k =  global_index / (xdim * ydim);

	jp = j;
	ip = (i < xdim) ? i : (i - ydim);
	kp = (k < xdim) ? k : (k - zdim);
	int ires = (kp * kp + ip * ip + jp * jp);
	if (global_index >= (xdim * ydim * zdim) || ires >= max_r2)
	{
		return;
	}
	int ires_id = ((sqrt((DOUBLE)ires) / padding_factor) > 0) ? (int)((sqrt((DOUBLE)ires) / padding_factor) + 0.5) : (int)((sqrt((DOUBLE)ires) / padding_factor) - 0.5);
	DOUBLE invw = oversampling_correction * weight_D[global_index];
#ifdef FLOAT_PRECISION
	atomicAdd(&(sigma2_D[ires_id]), (DOUBLE) invw);
	atomicAdd(&(counter_D[ires_id]), (DOUBLE) 1.);
#else
	atomicAdd_double(&(sigma2_D[ires_id]), (DOUBLE) invw);
	atomicAdd_double(&(counter_D[ires_id]), (DOUBLE) 1.);
#endif
	
}
__global__ void average_Sigma_kernel(DOUBLE* sigma2_D, DOUBLE* counter_D, int size_sigma)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (global_index >= size_sigma)
	{
		return;
	}
	sigma2_D[global_index] = (sigma2_D[global_index] > 1e-10) ? (counter_D[global_index] / sigma2_D[global_index]) : (0.);
}
void calculate_sigma2_gpu(DOUBLE* weight_D , DOUBLE* sigma2_D, DOUBLE* counter_D, int padding_factor, DOUBLE oversampling_correction,
                          int xdim, int ydim, int zdim, int max_r2, int size_sigma)
{
	int model_size = xdim * ydim * zdim;
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((model_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	calculate_sigma2_kernel <<< gridDim, blockDim>>>(weight_D, sigma2_D, counter_D,  padding_factor, oversampling_correction,
	                                                 xdim,  ydim,  zdim,  max_r2,  size_sigma);

	dim3 blockDim2(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim2((size_sigma + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	average_Sigma_kernel <<< gridDim2 , blockDim2>>>(sigma2_D, counter_D, size_sigma);
}
__global__ void init_Fconv_kernel(CUFFT_COMPLEX * Fconv_D, double* Fnewweight_D, DOUBLE* Fweight_D, int model_size)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if (global_index >= model_size)
	{
		return;
	}
	Fconv_D[global_index].x = Fnewweight_D[global_index] * Fweight_D[global_index];
	Fconv_D[global_index].y = 0.;
}

void init_Fconv_gpu(CUFFT_COMPLEX * Fconv_D, double* Fnewweight_D, DOUBLE* Fweight_D, long int model_size)
{
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((model_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);

	init_Fconv_kernel <<< gridDim, blockDim>>>(Fconv_D, Fnewweight_D, Fweight_D, model_size);

}
__global__ void Multi_by_FT_tab_kernel(DOUBLE* Mconv_D, const DOUBLE* __restrict__ tab_ftblob_D,  DOUBLE normftblob,  DOUBLE sampling, int pad_size, int padhdim, int ori_size_padding_factor, int padding_factor, int tab_size, bool do_mask)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (global_index >= (pad_size * pad_size * pad_size))
	{
		return;
	}

	int i, j, k;
	int ip, jp, kp;
	j = global_index % pad_size;
	i = (global_index / pad_size) % pad_size;
	k =  global_index / (pad_size * pad_size);
	kp = (k < padhdim) ? k : k - pad_size;
	ip = (i < padhdim) ? i : i - pad_size;
	jp = (j < padhdim) ? j : j - pad_size;
	DOUBLE rval = sqrt((DOUBLE)(kp * kp + ip * ip + jp * jp)) / (ori_size_padding_factor);
	if (do_mask && rval > 1. / (2. * padding_factor))
	{
		Mconv_D[global_index] = 0.;
	}
	else
	{
		int idx = (int)(abs(rval) / sampling);
		if (idx >= tab_size)
		{
			Mconv_D[global_index] = 0.;
		}
		else
		{
			Mconv_D[global_index] *= (tab_ftblob_D[idx] / normftblob);
		}
	}
}
void Multi_by_FT_tab_gpu(DOUBLE* Mconv_D,  DOUBLE* tab_ftblob_D,  DOUBLE normftblob,   DOUBLE sampling, int pad_size, int padhdim, int ori_size_padding_factor, int padding_factor, int tab_size, bool do_mask)
{
	int model_size = pad_size * pad_size * pad_size;
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((model_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	Multi_by_FT_tab_kernel <<< gridDim, blockDim>>>(Mconv_D, tab_ftblob_D, normftblob,  sampling, pad_size, padhdim, ori_size_padding_factor, padding_factor, tab_size, do_mask);
}

__global__ void update_Fconv_kernel(const CUFFT_COMPLEX * __restrict__ Fconv_D, double* Fnewweight_D, int xdim, int ydim, int zdim, int max_r2)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	int i, j, k;
	int ip, jp, kp;
	j = global_index % xdim;
	i = (global_index / xdim) % ydim;
	k =  global_index / (xdim * ydim);
	kp = (k < xdim) ? k : k - zdim;
	ip = (i < xdim) ? i : i - ydim;
	jp = j;
	int r2 = kp * kp + ip * ip + jp * jp;
	if (global_index >= (xdim * ydim * zdim) || r2 >= max_r2)
	{
		return;
	}
	DOUBLE w = sqrt(Fconv_D[global_index].x * Fconv_D[global_index].x + Fconv_D[global_index].y * Fconv_D[global_index].y);
	w = (((1e-6) >= w) ? (1e-6) : (w));
	Fnewweight_D[global_index] =Fnewweight_D[global_index]/w;

}
void update_Fconv_gpu(CUFFT_COMPLEX * Fconv_D, double* Fnewweight_D, int xdim, int ydim, int zdim, int max_r2)
{
	int model_size = xdim * ydim * zdim;
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((model_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	update_Fconv_kernel <<< gridDim , blockDim>>>(Fconv_D, Fnewweight_D, xdim, ydim, zdim, max_r2);
}



__global__ void centerFFT_2_kernel(DOUBLE* in, DOUBLE* out, int xdim, int ydim, int xshift, int yshift)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	int i, j;
	j = global_index % xdim;
	i = (global_index / xdim) % ydim;
	int jp = j + xshift;
	int ip = i + yshift;
	int posy = (ip >= ydim) ? (ip - ydim) : ((ip < 0) ? (ip + ydim) : ip);
	int posx = (jp >= xdim) ? (jp - xdim) : ((jp < 0) ? (jp + xdim) : jp);
	if (global_index >= xdim * ydim )
	{
		return;
	}

	out[posx + posy * xdim + blockIdx.y * xdim * ydim ] = in[j + i * xdim  + blockIdx.y * xdim * ydim];

}

//template <typename T>
__global__ void centerFFT_3_kernel(DOUBLE* in, DOUBLE* out, int xdim, int ydim, int zdim, int xshift, int yshift, int zshift)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	int i, j, k;
	j = global_index % xdim;
	i = (global_index / xdim) % ydim;
	k =  global_index / (xdim * ydim);
	int jp = j + xshift;
	int ip = i + yshift;
	int kp = k + zshift;
	int posz = (kp >= zdim) ? (kp - zdim) : ((kp < 0) ? (kp + zdim) : kp);
	int posy = (ip >= ydim) ? (ip - ydim) : ((ip < 0) ? (ip + ydim) : ip);
	int posx = (jp >= xdim) ? (jp - xdim) : ((jp < 0) ? (jp + xdim) : jp);
	if (global_index >= xdim * ydim * zdim)
	{
		return;
	}

	out[posx + posy * xdim + posz * xdim * ydim + blockIdx.y * xdim * ydim * zdim] = in[j + i * xdim + k * xdim * ydim + blockIdx.y * xdim * ydim * zdim];

}

//template <typename T>
void centerFFT_2_gpu(DOUBLE* in, DOUBLE* out, int nr_images, int dim, int xdim, int ydim, int zdim, bool forward)
{
	int size = xdim * ydim * zdim;

	dim3 dimBlock(BLOCK_SIZE_128, 1, 1);
	dim3 dimGrid((size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, nr_images, 1);
	int xshift = (int)(xdim / 2);
	int yshift = (int)(ydim / 2);
	int zshift = (int)(zdim / 2);
	if (!forward)
	{
		xshift = -xshift;
		yshift = -yshift;
		zshift = -zshift;
	}
	if(dim == 2)
		centerFFT_2_kernel <<< dimGrid, dimBlock>>>(in, out, xdim, ydim, xshift, yshift);
	else if(dim ==3)
		centerFFT_3_kernel <<< dimGrid, dimBlock>>>(in, out, xdim, ydim, zdim, xshift, yshift, zshift);
}

// Explicit instantiation
//template void centerFFT_2_gpu<DOUBLE>(DOUBLE* in, DOUBLE* out, int nr_images, int dim, int xdim, int ydim, int zdim, bool forward);
//template void centerFFT_2_gpu<float>(float* in, float* out, int nr_images, int dim, int xdim, int ydim, int zdim, bool forward);

//template <typename T>
__global__ void window_kernel(DOUBLE* in, DOUBLE* out, DOUBLE normfft, int start_x1,  int start_y1,  int start_z1,  int start_x2,  int start_y2,  int start_z2,
                              int x1dim, int y1dim, int z1dim, int x2dim, int y2dim, int z2dim,
                              DOUBLE init_value, int n)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (global_index >= x1dim * y1dim * z1dim)
	{
		return;
	}
	int i, j, k;
	j = global_index % x1dim + start_x1;
	i = (global_index / x1dim) % y1dim + start_y1;
	k =  global_index / (x1dim * y1dim) + start_z1;

	if (j >= start_x2 && j <= (start_x2 + x2dim - 1) && i >= start_y2 && i <= (start_y2 + y2dim - 1)  && k >= start_z2 && k <= (start_z2 + z2dim - 1))
	{
		out[global_index] = in[(k - start_z2) * x2dim * y2dim + (i - start_y2) * x2dim + (j - start_x2) + n * x2dim * y2dim * z2dim] / normfft;
	}
	else
	{
		out[global_index] = init_value / normfft;
	}

}
//template <typename T>
void window_gpu(DOUBLE* in, DOUBLE* out,  DOUBLE normfft, int start_x1,  int start_y1,  int start_z1,  int start_x2,  int start_y2,  int start_z2,
                int x1dim, int y1dim, int z1dim, int x2dim, int y2dim, int z2dim,
                DOUBLE init_value = 0, int n = 0)
{
	int data_size = (x1dim * y1dim * z1dim);

	dim3 dimBlock(BLOCK_SIZE_128, 1, 1);
	dim3 dimGrid((data_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	window_kernel <<< dimGrid , dimBlock>>>(in, out, normfft, start_x1, start_y1, start_z1, start_x2,   start_y2,   start_z2,
	                                        x1dim,  y1dim,  z1dim,  x2dim,  y2dim,  z2dim,
	                                        init_value,  n) ;

}
//template void window_gpu<DOUBLE>(DOUBLE* in, DOUBLE* out, DOUBLE normfft,  int start_x1,  int start_y1,  int start_z1,  int start_x2,  int start_y2,  int start_z2,
//                                 int x1dim, int y1dim, int z1dim, int x2dim, int y2dim, int z2dim,
//                                DOUBLE init_value = 0, int n = 0);
//template void window_gpu<float>(float* in, float* out, DOUBLE normfft, int start_x1,  int start_y1,  int start_z1,  int start_x2,  int start_y2,  int start_z2,
//                                int x1dim, int y1dim, int z1dim, int x2dim, int y2dim, int z2dim,
//                                float init_value = 0, int n = 0);

static __global__ void softMaskOutsideMap_new_kernel(DOUBLE* vol, DOUBLE radius, DOUBLE cosine_width, DOUBLE* Mnoise, DOUBLE radius_p, int xdim, int ydim, int zdim, int xinit, int yinit, int zinit)
{
	int offset;
	int tid = threadIdx.x;

	int image_size = zdim * ydim * xdim;

	if (tid >= image_size)
	{
		return;
	}
	long int kp, ip, jp;
	offset = blockIdx.x * image_size;
	DOUBLE r, raisedcos;

	__shared__ DOUBLE sum_bg[512];
	__shared__ DOUBLE sum[512];
	sum_bg[tid] = 0;
	sum[tid] = 0;

	if (Mnoise == NULL)
	{
		for (int i = tid; i < image_size; i += blockDim.x)
		{
			jp = ((i % xdim) + xinit);
			ip = ((i / xdim) % ydim + yinit);
			kp = (i / (xdim * ydim) + zinit);
			r = sqrt((DOUBLE)(kp * kp + ip * ip + jp * jp));

			if (r < radius)
				;
			else if (r > radius_p)
			{
				sum[tid]    += 1.;
				sum_bg[tid] += vol[offset + (kp - zinit) * xdim * ydim + (ip - yinit) * xdim + (jp - xinit)];
			}
			else
			{
				raisedcos = 0.5 + 0.5 * cos(PI * (radius_p - r) / cosine_width);
				sum[tid] += raisedcos;
				sum_bg[tid] += raisedcos * vol[offset + (kp - zinit) * xdim * ydim + (ip - yinit) * xdim + (jp - xinit)];
			}
		}
	}
	////////////TODO: refuction the sum of sum and sum_bg
	// do reduction in shared mem
	__syncthreads();
	for (unsigned int s = (blockDim.x / 2); s > 0; s = (s >> 1))
	{
		if (tid < s)
		{
			sum[tid] += sum[tid + s];
			sum_bg[tid] += sum_bg[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0)
	{
		sum_bg[0] /= sum[0];
	}
	__syncthreads();

	// Calculate average background value
	for (int i = tid; i < image_size; i += blockDim.x)
	{
		jp = (i % xdim + xinit);
		ip = (i / xdim) % ydim + yinit;
		kp = (i / (xdim * ydim) + zinit);
		r = sqrt((DOUBLE)(kp * kp + ip * ip + jp * jp));
		if (r > radius_p && r >= radius)
		{
			vol[offset + (kp - zinit)*xdim * ydim + (ip - yinit)*xdim + (jp - xinit)] = (Mnoise == NULL) ? sum_bg[0] : Mnoise[offset + (kp - zinit) * xdim * ydim + (ip - yinit) * xdim + (jp - xinit)];
		}
		else if (r <= radius_p && r >= radius)
		{
			raisedcos = 0.5 + 0.5 * cos(PI * (radius_p - r) / cosine_width);
			DOUBLE add = (Mnoise == NULL) ?  sum_bg[0] : Mnoise[offset + (kp - zinit) * xdim * ydim + (ip - yinit) * xdim + (jp - xinit)];
			vol[offset + (kp - zinit)*xdim * ydim + (ip - yinit)*xdim + (jp - xinit)] = (1 - raisedcos) * vol[offset + (kp - zinit) * xdim * ydim + (ip - yinit) * xdim + (jp - xinit)] + raisedcos * add;
		}
	}

}

//template <typename T>
void softMaskOutsideMap_new_gpu(DOUBLE* vol, DOUBLE radius, DOUBLE cosine_width, DOUBLE* Mnoise, int nr_images, int xdim, int ydim, int zdim)
{
	int zinit = FIRST_XMIPP_INDEX(zdim);
	int yinit = FIRST_XMIPP_INDEX(ydim);
	int xinit = FIRST_XMIPP_INDEX(xdim);

	DOUBLE radius_p;
	if (radius < 0)
	{
		radius = (DOUBLE)xdim / 2.;
	}
	radius_p = radius + cosine_width;

	dim3 blockDim(512, 1, 1);
	dim3 gridDim(nr_images, 1, 1);
	int shared_mem_size = 512 * sizeof(DOUBLE) * 2;

	softMaskOutsideMap_new_kernel <<< gridDim, blockDim>>>(vol,  radius,  cosine_width, Mnoise, radius_p, xdim, ydim, zdim, xinit, yinit, zinit);

}

//template void softMaskOutsideMap_new_gpu<DOUBLE>(DOUBLE* vol, DOUBLE radius, DOUBLE cosine_width, DOUBLE* Mnoise, int nr_images, int xdim, int ydim, int zdim);

extern __shared__ DOUBLE  spectrum_count[ ];
__global__ void update_tau2_kernel(CUFFT_COMPLEX * Fconv_D, DOUBLE* tau2_D, DOUBLE tau2_fudge, int xdim, int ydim, int zdim, int data_vs_prior_size, int ori_size, DOUBLE  normfft)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	DOUBLE* spectrum = (DOUBLE*)spectrum_count;
	DOUBLE* count = (DOUBLE*)&spectrum[ori_size];

	if (threadIdx.x < ori_size)
	{
		spectrum[threadIdx.x] = 0.;
		count[threadIdx.x] = 0.;
	}
	int i, j, k;
	int jp, ip, kp;
	for (global_index = threadIdx.x; global_index < xdim * ydim * zdim; global_index += blockDim.x)
	{
		j = ((global_index % xdim));
		i = ((global_index / xdim) % ydim);
		k = (global_index / (xdim * ydim));
		kp = (k < xdim) ? k : k - zdim;
		ip = (i < xdim) ? i : i - ydim;
		jp = j; // (j < padhdim) ? j : j - pad_size;
		int r2 = kp * kp + ip * ip + jp * jp;
		int idx = (sqrt((DOUBLE)r2)) > 0 ? (int)((sqrt((DOUBLE)r2)) + 0.5) : (int)((sqrt((DOUBLE)r2)) - 0.5);
		DOUBLE normmal = Fconv_D[global_index].x * Fconv_D[global_index].x + Fconv_D[global_index].y * Fconv_D[global_index].y;
#ifdef FLOAT_PRECISION
	atomicAdd(&(spectrum[idx]), (DOUBLE) normmal);
	atomicAdd(&(count[idx]), (DOUBLE) 1.);
#else
	atomicAdd_double(&(spectrum[idx]), (DOUBLE) normmal);
	atomicAdd_double(&(count[idx]), (DOUBLE) 1.);
#endif

	}
	__syncthreads();
	if (threadIdx.x < ori_size)
	{
		spectrum[threadIdx.x] /= count[threadIdx.x];
		spectrum[threadIdx.x] *= (normfft / 2.);
	}
	if (threadIdx.x  < data_vs_prior_size)
	{
		tau2_D[threadIdx.x] =  tau2_fudge * spectrum[threadIdx.x];
	}
}

void update_tau2_gpu(CUFFT_COMPLEX * Fconv_D, DOUBLE* tau2_D, DOUBLE tau2_fudge, int xdim, int ydim, int zdim, int data_vs_prior_size, int ori_size,  DOUBLE  normfft)
{

	dim3 dimBlock((ori_size >= 512) ? ori_size : 512, 1, 1);
	dim3 dimGrid(1, 1, 1);
	int shared_mem_size = sizeof(DOUBLE) * ori_size * 2;
	update_tau2_kernel <<< dimGrid, dimBlock, shared_mem_size>>>(Fconv_D, tau2_D, tau2_fudge, xdim, ydim, zdim, data_vs_prior_size, ori_size, normfft);


}

void BackProjector::reconstruct_gpu(MultidimArray<DOUBLE>& vol_out,
                                    int max_iter_preweight,
                                    bool do_map,
                                    DOUBLE tau2_fudge,
                                    MultidimArray<DOUBLE>& tau2,
                                    MultidimArray<DOUBLE>& sigma2,
                                    MultidimArray<DOUBLE>& data_vs_prior,
                                    MultidimArray<DOUBLE> fsc, // only input
                                    DOUBLE normalise,
                                    bool update_tau2_with_fsc,
                                    bool is_whole_instead_of_half,
                                    int nr_threads,
                                    int minres_map)

{


	FourierTransformer transformer;

	//MultidimArray<Complex > Fconv;
	//MultidimArray<DOUBLE> Fweight, Fnewweight;
	int max_r2 = r_max * r_max * padding_factor * padding_factor;

	size_t free, total;
	hipMemGetInfo(&free,  &total);
	std::cout << "GPU memor  y info total beginnig1 " << total / (1024 * 1024) << "MB  free  memory " << free / (1024 * 1024) << " MB "  << std::endl;
	// At the x=0 line, we have collected either the positive y-z coordinate, or its negative Friedel pair.
	// Sum these two together for both the data and the weight arrays
	CUFFT_COMPLEX * data_D;
	DOUBLE* weight_D;
	int xdim = data.xdim;
	int ydim = data.ydim;
	int xydim = data.yxdim;
	int zdim = data.zdim;
	int start_x = STARTINGX(data);
	int start_y = STARTINGY(data);
	int start_z = STARTINGZ(data);
	hipMalloc((void**)&data_D, data.zyxdim * sizeof(CUFFT_COMPLEX ));
	hipMalloc((void**)&weight_D, data.zyxdim * sizeof(DOUBLE));
	hipMemcpy(data_D, data.data, data.zyxdim * sizeof(CUFFT_COMPLEX ), hipMemcpyHostToDevice);
	hipMemcpy(weight_D, weight.data, data.zyxdim * sizeof(DOUBLE), hipMemcpyHostToDevice);
	if (data.zdim > 1)
	{
		enforceHermitianSymmetry_gpu(data_D - start_x, weight_D - start_x, xdim, ydim, xydim, zdim);
	}

	hipError_t cudaStat = hipGetLastError();
	if (cudaStat != hipSuccess)
	{
		printf("kernel symmetrise_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
		exit(EXIT_FAILURE);
	}
	symmetrise_gpu(data_D,
	               weight_D,
	               xdim,
	               ydim,
	               xydim,
	               zdim,
	               start_x,
	               start_y,
	               start_z,
	               max_r2);
	cudaStat = hipGetLastError();
	if (cudaStat != hipSuccess)
	{
		printf("kernel symmetrise_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
		exit(EXIT_FAILURE);
	}

	int new_xdim = pad_size / 2 + 1;
	int new_ydim = pad_size;
	int new_zdim = (ref_dim == 2) ? 1 : pad_size;
	int new_model_size = new_xdim * new_ydim * new_zdim;
	DOUBLE* Fweight_D;
       // Fnewweight can become too large for a float: always keep this one in double-precision
	double *Fnewweight_D;
	DOUBLE* sigma2_D,  *counter_D;
	CUFFT_COMPLEX * Fconv_D;
	DOUBLE* vol_out_D;

	/*if (ref_dim == 2)
	{
		hipMalloc((void**)&Fconv_D, (pad_size / 2 + 1)*pad_size * sizeof(DOUBLE) * 2);
	}
	else
	{
		hipMalloc((void**)&Fconv_D, (pad_size / 2 + 1)*pad_size * pad_size * sizeof(DOUBLE) * 2);
	}
*/
	cudaStat = hipGetLastError();
	if (cudaStat != hipSuccess)
	{
		printf("kernel symmetrise_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
		exit(EXIT_FAILURE);
	}
	// clear vol_out to save memory!

	// Take oversampling into account
	DOUBLE oversampling_correction = (ref_dim == 3) ? (padding_factor * padding_factor * padding_factor) : (padding_factor * padding_factor);

	// First calculate the radial average of the (inverse of the) power of the noise in the reconstruction
	// This is the left-hand side term in the nominator of the Wiener-filter-like update formula
	// and it is stored inside the weight vector
	// Then, if (do_map) add the inverse of tau2-spectrum values to the weight
	hipMalloc((void**) & sigma2_D, (ori_size / 2 + 1)*sizeof(DOUBLE));
	hipMalloc((void**) & counter_D, (ori_size / 2 + 1)*sizeof(DOUBLE));
	hipMemset(sigma2_D, 0., (ori_size / 2 + 1)*sizeof(DOUBLE));
	hipMemset(counter_D, 0., (ori_size / 2 + 1)*sizeof(DOUBLE));
	hipMalloc((void**) & Fweight_D, new_model_size * sizeof(DOUBLE));
	hipMalloc((void**) & Fnewweight_D, new_model_size * sizeof(double));

	hipMemset(Fweight_D, 0., new_model_size * sizeof(DOUBLE));
	hipMemset(Fnewweight_D, 0., new_model_size * sizeof(double));

	decenter_gpu(weight_D,
	             Fweight_D,
	             max_r2,
	             new_xdim,
	             new_ydim,
	             new_zdim,
	             xdim,
	             ydim,
	             start_x,
	             start_y,
	             start_z);

	calculate_sigma2_gpu(
	    Fweight_D,
	    sigma2_D,
	    counter_D,
	    padding_factor,
	    oversampling_correction,
	    new_xdim,
	    new_ydim,
	    new_zdim, max_r2,
	    ori_size / 2 + 1);

	DOUBLE* fsc_D, *tau2_D, *data_vs_prior_D;
	hipMalloc((void**) &fsc_D, fsc.xdim * sizeof(DOUBLE));
	hipMalloc((void**) &tau2_D, tau2.xdim * sizeof(DOUBLE));
	hipMalloc((void**) &data_vs_prior_D, (ori_size / 2 + 1)*sizeof(DOUBLE));
	hipMemcpy(fsc_D, fsc.data, fsc.xdim * sizeof(DOUBLE), hipMemcpyHostToDevice);
	hipMemcpy(tau2_D, tau2.data, tau2.xdim * sizeof(DOUBLE), hipMemcpyHostToDevice);
	hipMemset(data_vs_prior_D, 0., (ori_size / 2 + 1)*sizeof(DOUBLE));

	//Due to the value of (ori_size/2 + 1) is very limited, we remain the follow section to be processed in CPU side
	if (update_tau2_with_fsc)
	{
		update_tau2_with_fsc_gpu(sigma2_D, fsc_D, tau2_D, data_vs_prior_D, (ori_size / 2 + 1), is_whole_instead_of_half);
	}

	if (do_map)
	{
		if (!update_tau2_with_fsc)
		{
			hipMemset(data_vs_prior_D, 0., (ori_size / 2 + 1)*sizeof(DOUBLE));
		}
		hipMemset(counter_D, 0., (ori_size / 2 + 1)*sizeof(DOUBLE));
		Applymap_additional_to_weight_gpu(Fweight_D, tau2_D, data_vs_prior_D, counter_D,
		                                  max_r2, new_xdim, new_ydim, new_zdim,
		                                  padding_factor, oversampling_correction, tau2_fudge, update_tau2_with_fsc, minres_map);
		if (!update_tau2_with_fsc)
		{
			Average_data_vs_prior_gpu(data_vs_prior_D, counter_D, ori_size / 2 + 1, r_max);
		}

	}

	// Divide both data and Fweight by normalisation factor to prevent FFT's with very large values....
	//std::cout <<"Running the GPU Fweight_D  after:" << data.zdim << std::endl;
	do_normalise_weight_data_gpu(Fweight_D, data_D, new_model_size, data.zyxdim, normalise);
	init_Fnewweight_gpu(Fnewweight_D, new_xdim, new_ydim, new_zdim, max_r2);


	int tab_size = tab_ftblob.tabulatedValues.xdim;
	DOUBLE* tabulatedValues_D;
	hipMalloc((void**)&tabulatedValues_D, tab_size * sizeof(DOUBLE));
	hipMemcpy(tabulatedValues_D, tab_ftblob.tabulatedValues.data, tab_size * sizeof(DOUBLE), hipMemcpyHostToDevice);
	// Iterative algorithm as in  Eq. [14] in Pipe & Menon (1999)
	// or Eq. (4) in Matej (2001)
	hipfftHandle fPlanForward_gpu;
	hipfftHandle fPlanBackward_gpu;
	hipfftResult fftplan1, fftplan2;
//	size_t free, total;
	hipMemGetInfo(&free,  &total);
	std::cout << "GPU memory info total 1 " << total / (1024 * 1024) << "MB  free  memory " << free / (1024 * 1024) << " MB "  << std::endl;
	std::cout << "cufft need memory " << pad_size * pad_size * (ref_dim == 2? 1: pad_size) << std::endl;

	//std::cout << "The fft plan size is " << pad_size* pad_size* (ref_dim == 2 ? 1 : pad_size) << " paded " << pad_size << std::endl;
#ifdef FLOAT_PRECISION
	fftplan1 = hipfftPlan3d(&fPlanBackward_gpu ,  pad_size, pad_size, (ref_dim == 2 ? 1 : pad_size), HIPFFT_C2R);
#else
	fftplan1 = hipfftPlan3d(&fPlanBackward_gpu ,  pad_size, pad_size, (ref_dim == 2 ? 1 : pad_size), HIPFFT_Z2D);
#endif
	
	if (fPlanBackward_gpu == NULL)
	{
		std::cerr << " fftplan create failed fPlanBackward_gpu= " << fftplan1 << " fPlanBackward= "   << " iter " << pad_size << std::endl;
	}
#ifdef FLOAT_PRECISION
		fftplan2 = hipfftPlan3d(&fPlanForward_gpu ,  pad_size, pad_size, (ref_dim == 2 ? 1 : pad_size), HIPFFT_R2C);
#else
		fftplan2 = hipfftPlan3d(&fPlanForward_gpu ,  pad_size, pad_size, (ref_dim == 2 ? 1 : pad_size), HIPFFT_D2Z);
#endif
	
	if (fPlanForward_gpu == NULL)
	{
		std::cerr << " fftplan create failed fPlanForward_gpu= " << fftplan2 << " fPlanForward_gpu= "    << " iter " << ref_dim << std::endl;
	}
	DOUBLE* Mconv_D;
	if (ref_dim == 2)
	{
		hipMalloc((void**)&Mconv_D, pad_size * pad_size * sizeof(DOUBLE));
		hipMalloc((void**)&Fconv_D, (pad_size / 2 + 1)*pad_size * sizeof(DOUBLE) * 2);
		hipMemset(Mconv_D, 0., pad_size * pad_size * sizeof(DOUBLE));
		hipMemset(Fconv_D, 0., (pad_size / 2 + 1)*pad_size * sizeof(DOUBLE) * 2);
	}
	else
	{
		hipMalloc((void**)&Mconv_D, pad_size * pad_size * pad_size * sizeof(DOUBLE));
		hipMalloc((void**)&Fconv_D, (pad_size / 2 + 1)*pad_size * pad_size * sizeof(DOUBLE) * 2);
		hipMemset(Mconv_D, 0., pad_size * pad_size * pad_size * sizeof(DOUBLE));
		hipMemset(Fconv_D, 0., (pad_size / 2 + 1)*pad_size * pad_size * sizeof(DOUBLE) * 2);
	}

	for (int iter = 0; iter < max_iter_preweight; iter++)
	{

		init_Fconv_gpu(Fconv_D, Fnewweight_D, Fweight_D, new_model_size);
		//======================================================
		DOUBLE normftblob = tab_ftblob(0.);
#ifdef FLOAT_PRECISION
		hipfftExecC2R(fPlanBackward_gpu,  Fconv_D, Mconv_D);
#else
	       hipfftExecZ2D(fPlanBackward_gpu,  Fconv_D, Mconv_D);
#endif

		//transformer.inverseFourierTransform_gpu(Fconv_D, Mconv_D, 1, pad_size, pad_size, (ref_dim==2?1:pad_size));
		cudaStat = hipGetLastError();
		if (cudaStat != hipSuccess)
		{
			printf("kernel symmetrise_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
			exit(EXIT_FAILURE);
		}
		Multi_by_FT_tab_gpu(Mconv_D, tabulatedValues_D,  tab_ftblob(0.),  tab_ftblob.sampling, pad_size, pad_size / 2, ori_size * padding_factor, padding_factor, tab_ftblob.tabulatedValues.xdim, false);
#ifdef FLOAT_PRECISION
		hipfftExecR2C(fPlanForward_gpu,  Mconv_D, Fconv_D);
#else
	       hipfftExecD2Z(fPlanForward_gpu,  Mconv_D, Fconv_D);
#endif

		ScaleComplexPointwise_gpu(Fconv_D, (pad_size / 2 + 1)*pad_size * (ref_dim == 2 ? 1 : pad_size), 1.0 / (pad_size * pad_size * (ref_dim == 2 ? 1 : pad_size)));

		cudaStat = hipGetLastError();
		if (cudaStat != hipSuccess)
		{
			printf("kernel symmetrise_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
			exit(EXIT_FAILURE);
		}
		update_Fconv_gpu(Fconv_D, Fnewweight_D, new_xdim, new_ydim, new_zdim,  max_r2);

	}
	hipfftDestroy(fPlanForward_gpu);
	hipfftDestroy(fPlanBackward_gpu);
	hipMemset(Fconv_D, 0., new_model_size * sizeof(CUFFT_COMPLEX ));

	decenter_gpu(data_D,
	             Fconv_D,
	             Fnewweight_D,
	             max_r2,
	             new_xdim,
	             new_ydim,
	             new_zdim,
	             xdim,
	             ydim,
	             start_x,
	             start_y,
	             start_z);

	tau2.initZeros(ori_size / 2 + 1);
	data_vs_prior.initZeros(ori_size / 2 + 1);
	sigma2.initZeros(ori_size / 2 + 1);
	if (update_tau2_with_fsc)
	{
		hipMemcpy(tau2.data, tau2_D, (ori_size / 2 + 1)*sizeof(DOUBLE), hipMemcpyDeviceToHost);
	}
	hipMemcpy(sigma2.data, sigma2_D, (ori_size / 2 + 1)*sizeof(DOUBLE), hipMemcpyDeviceToHost);
	hipMemcpy(data_vs_prior.data, data_vs_prior_D, (ori_size / 2 + 1)*sizeof(DOUBLE), hipMemcpyDeviceToHost);

	hipMemcpy(data.data, data_D, data.zyxdim * sizeof(CUFFT_COMPLEX ), hipMemcpyDeviceToHost);
	hipMemcpy(weight.data, weight_D, data.zyxdim * sizeof(DOUBLE), hipMemcpyDeviceToHost);


	// Now do inverse FFT and window to original size in real-space
	// Pass the transformer to prevent making and clearing a new one before clearing the one declared above....
	// The latter may give memory problems as detected by electric fence....
	//windowToOridimRealSpace(transformer, Fconv, vol_out, nr_threads);
	int padoridim = padding_factor * ori_size;
	if (ref_dim == 2)
	{
		hipMalloc((void**)&vol_out_D, padoridim * padoridim * sizeof(DOUBLE)); //Mout.resize(padoridim, padoridim);
		hipMemset(vol_out_D, 0., padoridim * padoridim * sizeof(DOUBLE));

	}
	else
	{
		hipMalloc((void**)&vol_out_D, padoridim * padoridim * padoridim * sizeof(DOUBLE));
		hipMemset(vol_out_D, 0., padoridim * padoridim * padoridim * sizeof(DOUBLE));
	}
	//release some memory
	hipFree(data_D);
	hipFree(weight_D);
	hipFree(Fweight_D);
	hipFree(Fnewweight_D);
	hipFree(tabulatedValues_D);
	hipFree(Mconv_D);
	windowToOridimRealSpace_gpu(transformer, Fconv_D,
	                            vol_out_D,
	                            new_xdim,
	                            new_ydim,
	                            new_zdim);

	// Correct for the linear/nearest-neighbour interpolation that led to the data array
	griddingCorrect_gpu(vol_out_D,  ori_size, ori_size, ori_size,  interpolator,  r_min_nn,  ori_size * padding_factor);
	vol_out.resize((ref_dim == 2 ? 1 : ori_size), ori_size, ori_size);
	vol_out.setXmippOrigin();
	hipMemcpy(vol_out.data, vol_out_D, (ori_size * ori_size * (ref_dim == 2 ? 1 : ori_size)*sizeof(DOUBLE)), hipMemcpyDeviceToHost);

	cudaStat = hipGetLastError();
	if (cudaStat != hipSuccess)
	{
		printf("kernel calculate_weight_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
		exit(EXIT_FAILURE);
	}
	
	hipFree(Fconv_D);
	// If the tau-values were calculated based on the FSC, then now re-calculate the power spectrum of the actual reconstruction
	if (update_tau2_with_fsc)
	{

		DOUBLE* temp;
		hipMalloc((void**)&temp, (ori_size * ori_size * (ref_dim == 2 ? 1 : ori_size)*sizeof(DOUBLE)));

		hipMemcpy(temp, vol_out_D, (ori_size * ori_size * (ref_dim == 2 ? 1 : ori_size)*sizeof(DOUBLE)), hipMemcpyDeviceToDevice);
		hipMalloc((void**)&Fconv_D, ((ori_size / 2 + 1) * ori_size * (ref_dim == 2 ? 1 : ori_size)*sizeof(DOUBLE) * 2));
#ifdef FLOAT_PRECISION
		fftplan1 = hipfftPlan3d(&fPlanForward_gpu ,  ori_size, ori_size, (ref_dim == 2 ? 1 : ori_size), HIPFFT_R2C);
#else
		fftplan1 = hipfftPlan3d(&fPlanForward_gpu ,  ori_size, ori_size, (ref_dim == 2 ? 1 : ori_size), HIPFFT_D2Z);
#endif
		if (fPlanForward_gpu == NULL)
		{
			std::cerr << " fftplan create failed fPlanBackward_gpu= " << fftplan1 << " fPlanBackward= "   << std::endl;
			REPORT_ERROR("CUFFT Error: Unable to create plan");
		}
#ifdef FLOAT_PRECISION
		hipfftExecR2C(fPlanForward_gpu,  temp, Fconv_D);
#else
		hipfftExecD2Z(fPlanForward_gpu,  temp, Fconv_D);
#endif
		ScaleComplexPointwise_gpu(Fconv_D, (ori_size / 2 + 1)*ori_size * (ref_dim == 2 ? 1 : ori_size), 1.0 / (ori_size * ori_size * (ref_dim == 2 ? 1 : ori_size)));
		hipfftDestroy(fPlanForward_gpu);

		cudaStat = hipGetLastError();
		if (cudaStat != hipSuccess)
		{
			printf("kernel calculate_weight_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
			exit(EXIT_FAILURE);
		}
		update_tau2_gpu(Fconv_D, tau2_D, tau2_fudge, (ori_size / 2 + 1) , ori_size, (ref_dim == 2 ? 1 : ori_size), (ori_size / 2 + 1) , ori_size, (ref_dim == 3) ? (DOUBLE)(ori_size * ori_size) : 1.);
		hipMemcpy(tau2.data, tau2_D, (ori_size / 2 + 1)*sizeof(DOUBLE), hipMemcpyDeviceToHost);
		hipFree(Fconv_D);
		hipFree(temp);

	}
	cudaStat = hipGetLastError();
	if (cudaStat != hipSuccess)
	{
		printf("kernel calculate_weight_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
		exit(EXIT_FAILURE);
	}
	// Completely empty the transformer object

	hipFree(sigma2_D);
	hipFree(counter_D);
	hipFree(fsc_D);
	hipFree(tau2_D);
	hipFree(data_vs_prior_D);
	hipFree(vol_out_D);

	//transformer.free_memory_gpu();
	transformer.cleanup();
	cudaStat = hipGetLastError();
	if (cudaStat != hipSuccess)
	{
		printf("kernel calculate_weight_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
		exit(EXIT_FAILURE);
	}
	hipMemGetInfo(&free,  &total);
	std::cout << "GPU memor  y info total end " << total / (1024 * 1024) << "MB  free  memory " << free / (1024 * 1024) << " MB "  << std::endl;
}

__global__ void enforceHermitianSymmetry_kernel(CUFFT_COMPLEX * my_data,
                                                DOUBLE* my_weight,
                                                int xdim,
                                                int ydim,
                                                int xydim,
                                                int zdim
                                               )
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	int yindex = global_index % ydim;
	int zindex = global_index / ydim;
	if (zindex > (zdim / 2))
	{
		return;
	}

	DOUBLE real1, real2, img1, img2;
	DOUBLE sum;
	if (zindex == (zdim / 2))
	{
		if (yindex < (ydim / 2))
		{
			real1 = my_data[zindex * xydim + yindex * xdim].x;
			img1 = my_data[zindex * xydim + yindex * xdim].y;
			real2 = my_data[((zdim - 1) - zindex) * xydim + (ydim - 1 - yindex) * xdim].x;
			img2 = my_data[((zdim - 1) - zindex) * xydim + (ydim - 1 - yindex) * xdim].y;

			my_data[zindex * xydim + yindex * xdim].x = real1 + real2;
			my_data[zindex * xydim + yindex * xdim].y = img1 - img2;
			my_data[((zdim - 1) - zindex)*xydim + (ydim - 1 - yindex)*xdim].x = real1 + real2;
			my_data[((zdim - 1) - zindex)*xydim + (ydim - 1 - yindex)*xdim].y = img2 - img1;

			sum = my_weight[zindex * xydim + yindex * xdim] + my_weight[(zdim - 1 - zindex) * xydim + (ydim - 1 - yindex) * xdim];
			my_weight[zindex * xydim + yindex * xdim] = sum;
			my_weight[((zdim - 1) - zindex)*xydim + (ydim - 1 - yindex)*xdim] = sum;
		}
	}
	else
	{
		real1 = my_data[zindex * xydim + yindex * xdim].x;
		img1 = my_data[zindex * xydim + yindex * xdim].y;
		real2 = my_data[((zdim - 1) - zindex) * xydim + (ydim - 1 - yindex) * xdim].x;
		img2 = my_data[((zdim - 1) - zindex) * xydim + (ydim - 1 - yindex) * xdim].y;

		my_data[zindex * xydim + yindex * xdim].x = real1 + real2;
		my_data[zindex * xydim + yindex * xdim].y = img1 - img2;
		my_data[((zdim - 1) - zindex)*xydim + (ydim - 1 - yindex)*xdim].x = real1 + real2;
		my_data[((zdim - 1) - zindex)*xydim + (ydim - 1 - yindex)*xdim].y = img2 - img1;

		sum = my_weight[zindex * xydim + yindex * xdim] + my_weight[((zdim - 1) - zindex) * xydim + (ydim - 1 - yindex) * xdim];
		my_weight[zindex * xydim + yindex * xdim] = sum;
		my_weight[((zdim - 1) - zindex)*xydim + (ydim - 1 - yindex)*xdim] = sum;
	}

}
void BackProjector::enforceHermitianSymmetry_gpu(CUFFT_COMPLEX * my_data_D,
                                                 DOUBLE* my_weight_D,
                                                 int xdim,
                                                 int ydim,
                                                 int xydim,
                                                 int zdim)
{
	int nr_pair_points = ydim * ((zdim + 1) / 2);
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((nr_pair_points + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);

	enforceHermitianSymmetry_kernel <<< gridDim, blockDim>>>(my_data_D,
	                                                         my_weight_D,
	                                                         xdim,
	                                                         ydim,
	                                                         xydim,
	                                                         zdim);

}

__global__ void symmetrise_kernel(const CUFFT_COMPLEX * __restrict__ my_data_temp_D ,
                                  const  DOUBLE* __restrict__ my_weight_temp_D,
                                  CUFFT_COMPLEX * my_data_D,
                                  DOUBLE* my_weight_D,
                                  int xdim,
                                  int ydim,
                                  int xydim,
                                  int zdim,
                                  int start_x,
                                  int start_y,
                                  int start_z,
                                  int my_rmax2,
                                  int nr_SymsNo)

{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	int x, y, z;
	x = global_index % xdim + start_x;
	y = (global_index / xdim) % ydim + start_y;
	z =  global_index / xydim + start_z;
	if ((x * x + y * y + z * z) > my_rmax2 || global_index >= xydim * zdim)
	{
		return;
	}

	DOUBLE  fx, fy, fz, xp, yp, zp;
	bool is_neg_x;
	int x0, x1, y0, y1, z0, z1;
	DOUBLE d000_r, d001_r, d010_r, d011_r, d100_r, d101_r, d110_r, d111_r;
	DOUBLE dx00_r, dx01_r, dx10_r, dx11_r, dxy0_r, dxy1_r;
	DOUBLE d000_i, d001_i, d010_i, d011_i, d100_i, d101_i, d110_i, d111_i;
	DOUBLE dx00_i, dx01_i, dx10_i, dx11_i, dxy0_i, dxy1_i;
	DOUBLE dd000, dd001, dd010, dd011, dd100, dd101, dd110, dd111;
	DOUBLE ddx00, ddx01, ddx10, ddx11, ddxy0, ddxy1;

	DOUBLE real, img, weight;
	weight = real = img = 0.;

	for (int i = 0; i < nr_SymsNo; i++)
	{
		// coords_output(x,y) = A * coords_input (xp,yp)

		xp = (DOUBLE)x * __R_array[i * 4 * 4] + (DOUBLE)y *  __R_array[i * 4 * 4 + 1] + (DOUBLE)z *  __R_array[i * 4 * 4 + 2];
		yp = (DOUBLE)x *  __R_array[i * 4 * 4 + 1 * 4] + (DOUBLE)y *  __R_array[i * 4 * 4 + 1 + 1 * 4] + (DOUBLE)z *  __R_array[i * 4 * 4 + 2 + 1 * 4];
		zp = (DOUBLE)x *  __R_array[i * 4 * 4 + 2 * 4] + (DOUBLE)y *  __R_array[i * 4 * 4 + 1 + 2 * 4] + (DOUBLE)z *  __R_array[i * 4 * 4 + 2 + 2 * 4];
		// Only asymmetric half is stored
		if (xp < 0)
		{
			// Get complex conjugated hermitian symmetry pair
			xp = -xp;
			yp = -yp;
			zp = -zp;
			is_neg_x = true;
		}
		else
		{
			is_neg_x = false;
		}

		// Trilinear interpolation (with physical coords)
		// Subtract STARTINGY and STARTINGZ to accelerate access to data (STARTINGX=0)
		// In that way use DIRECT_A3D_ELEM, rather than A3D_ELEM
		x0 = floor(xp);
		fx = xp - x0;
		x1 = x0 + 1;

		y0 = floor(yp);
		fy = yp - y0;
		y0 -=  start_y;
		y1 = y0 + 1;

		z0 = floor(zp);
		fz = zp - z0;
		z0 -= start_z;
		z1 = z0 + 1;

		// First interpolate (complex) data
		d000_r = my_data_temp_D[z0 * xydim + y0 * xdim + x0].x;
		d001_r = my_data_temp_D[z0 * xydim + y0 * xdim + x1].x;
		d010_r = my_data_temp_D[z0 * xydim + y1 * xdim + x0].x;
		d011_r = my_data_temp_D[z0 * xydim + y1 * xdim + x1].x;

		d000_i = my_data_temp_D[z0 * xydim + y0 * xdim + x0].y;
		d001_i = my_data_temp_D[z0 * xydim + y0 * xdim + x1].y;
		d010_i = my_data_temp_D[z0 * xydim + y1 * xdim + x0].y;
		d011_i = my_data_temp_D[z0 * xydim + y1 * xdim + x1].y;

		d100_r = my_data_temp_D[z1 * xydim + y0 * xdim + x0].x;
		d101_r = my_data_temp_D[z1 * xydim + y0 * xdim + x1].x;
		d110_r = my_data_temp_D[z1 * xydim + y1 * xdim + x0].x;
		d111_r = my_data_temp_D[z1 * xydim + y1 * xdim + x1].x;

		d100_i = my_data_temp_D[z1 * xydim + y0 * xdim + x0].y;
		d101_i = my_data_temp_D[z1 * xydim + y0 * xdim + x1].y;
		d110_i = my_data_temp_D[z1 * xydim + y1 * xdim + x0].y;
		d111_i = my_data_temp_D[z1 * xydim + y1 * xdim + x1].y;

		dx00_r = d000_r + (d001_r - d000_r) * fx;
		dx00_i = d000_i + (d001_i - d000_i) * fx;
		dx01_r = d100_r + (d101_r - d100_r) * fx;
		dx01_i = d100_i + (d101_i - d100_i) * fx;
		dx10_r = d010_r + (d011_r - d010_r) * fx;
		dx10_i = d010_i + (d011_i - d010_i) * fx;
		dx11_r = d110_r + (d111_r - d110_r) * fx;
		dx11_i = d110_i + (d111_i - d110_i) * fx;

		dxy0_r = dx00_r + (dx10_r - dx00_r) * fy;
		dxy0_i = dx00_i + (dx10_i - dx00_i) * fy;
		dxy1_r = dx01_r + (dx11_r - dx01_r) * fy;
		dxy1_i = dx01_i + (dx11_i - dx01_i) * fy;
		if (is_neg_x)
		{
			real += dxy0_r + (dxy1_r - dxy0_r) * fz;
			img -= (dxy0_i + (dxy1_i - dxy0_i) * fz);
		}
		else
		{
			real += dxy0_r + (dxy1_r - dxy0_r) * fz;
			img += (dxy0_i + (dxy1_i - dxy0_i) * fz);
		}

		// Then interpolate (real) weight
		dd000 = my_weight_temp_D[z0 * xydim + y0 * xdim + x0];
		dd001 = my_weight_temp_D[z0 * xydim + y0 * xdim + x1];
		dd010 = my_weight_temp_D[z0 * xydim + y1 * xdim + x0];
		dd011 = my_weight_temp_D[z0 * xydim + y1 * xdim + x1];
		dd100 = my_weight_temp_D[z1 * xydim + y0 * xdim + x0];
		dd101 = my_weight_temp_D[z1 * xydim + y0 * xdim + x1];
		dd110 = my_weight_temp_D[z1 * xydim + y1 * xdim + x0];
		dd111 = my_weight_temp_D[z1 * xydim + y1 * xdim + x1];

		ddx00 = dd000 + (dd001 - dd000) * fx;
		ddx01 = dd100 + (dd101 - dd100) * fx;
		ddx10 = dd010 + (dd011 - dd010) * fx;
		ddx11 = dd110 + (dd111 - dd110) * fx;
		ddxy0 = ddx00 + (ddx10 - ddx00) * fy;
		ddxy1 = ddx01 + (ddx11 - ddx01) * fy;
		weight += ddxy0 + (ddxy1 - ddxy0) * fz;

	}
	my_data_D[global_index].x += real;
	my_data_D[global_index].y += img;
	my_weight_D[global_index] += weight;
}
void BackProjector::symmetrise_gpu(CUFFT_COMPLEX * my_data_D,
                                   DOUBLE* my_weight_D,
                                   int xdim,
                                   int ydim,
                                   int xydim,
                                   int zdim,
                                   int start_x,
                                   int start_y,
                                   int start_z,
                                   int my_rmax2
                                  )
{
	if (SL.SymsNo() > 0 && ref_dim == 3)
	{
		int model_size = xydim * zdim;
		DOUBLE* my_weight_temp_D;
		CUFFT_COMPLEX * my_data_temp_D;
		hipMemcpyToSymbol(HIP_SYMBOL(__L_array), SL.__L.mdata, SL.SymsNo() * 4 * 4 * sizeof(DOUBLE), 0 , hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(__R_array), SL.__R.mdata, SL.SymsNo() * 4 * 4 * sizeof(DOUBLE), 0 , hipMemcpyHostToDevice);
		dim3 blockDim(BLOCK_SIZE_128, 1, 1);
		dim3 gridDim((model_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);

		hipMalloc((void**)&my_weight_temp_D, model_size * sizeof(DOUBLE));
		hipMalloc((void**)&my_data_temp_D, model_size * sizeof(CUFFT_COMPLEX ));
		hipMemcpy(my_data_temp_D, my_data_D, model_size * sizeof(CUFFT_COMPLEX ), hipMemcpyDeviceToDevice);
		hipMemcpy(my_weight_temp_D, my_weight_D, model_size * sizeof(DOUBLE), hipMemcpyDeviceToDevice);
		hipError_t cudaStat = hipGetLastError();
		if (cudaStat != hipSuccess)
		{
			printf("kernel symmetrise_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
			exit(EXIT_FAILURE);
		}
		symmetrise_kernel <<< gridDim, blockDim >>>(my_data_temp_D,
		                                            my_weight_temp_D,
		                                            my_data_D,
		                                            my_weight_D,
		                                            xdim,
		                                            ydim,
		                                            xydim,
		                                            zdim,
		                                            start_x,
		                                            start_y,
		                                            start_z,
		                                            my_rmax2,
		                                            SL.SymsNo());
		hipFree(my_data_temp_D);
		hipFree(my_weight_temp_D);
	}

}

__global__ void decenter_kernel(const DOUBLE* __restrict__  weight_D, DOUBLE* Fweight_D, int max_r2,
                                int xdim, int ydim, int zdim, int xdim_weight, int ydim_weight,
                                int start_x, int start_y, int start_z)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	int i, j, k;
	int ip, jp, kp;
	j = global_index % xdim;
	i = (global_index / xdim) % ydim;
	k =  global_index / (xdim * ydim);

	jp = j;
	ip = (i < xdim) ? i : (i - ydim);
	kp = (k < xdim) ? k : (k - zdim);
	int ires = (kp * kp + ip * ip + jp * jp);
	if (global_index >= (xdim * ydim * zdim) || ires > max_r2)
	{
		return;
	}

	Fweight_D[global_index] = weight_D[(kp - start_z) * xdim_weight * ydim_weight + (ip - start_y) * xdim_weight + jp - start_x];

}

void BackProjector::decenter_gpu(DOUBLE* weight_D, DOUBLE* Fweight_D, int max_r2,
                                 int xdim, int ydim, int zdim, int xdim_weight, int ydim_weight,
                                 int start_x, int start_y, int start_z)
{
	int model_size = xdim * ydim * zdim;
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((model_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	decenter_kernel <<< gridDim, blockDim>>>(weight_D, Fweight_D, max_r2,
	                                         xdim, ydim, zdim, xdim_weight, ydim_weight,
	                                         start_x, start_y, start_z);

}

__global__ void decenter_kernel(const CUFFT_COMPLEX * __restrict__ data_D, CUFFT_COMPLEX * Fconv_D, const double* __restrict__ Fnewweight_D, int max_r2,
                                int xdim, int ydim, int zdim, int xdim_weight, int ydim_weight,
                                int start_x, int start_y, int start_z)
{
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	int i, j, k;
	int ip, jp, kp;
	j = global_index % xdim;
	i = (global_index / xdim) % ydim;
	k =  global_index / (xdim * ydim);

	jp = j;
	ip = (i < xdim) ? i : (i - ydim);
	kp = (k < xdim) ? k : (k - zdim);
	int ires = (kp * kp + ip * ip + jp * jp);
	if (global_index >= (xdim * ydim * zdim) || ires > max_r2)
	{
		return;
	}
	double Fnewweight = Fnewweight_D[global_index];
#ifdef  FLOAT_PRECISION
            // Prevent numerical instabilities in single-precision reconstruction with very unevenly sampled orientations
            if (Fnewweight > 1e20)
                Fnewweight = 1e20;
#endif
	Fconv_D[global_index].x = data_D[(kp - start_z) * xdim_weight * ydim_weight + (ip - start_y) * xdim_weight + jp - start_x].x * Fnewweight;
	Fconv_D[global_index].y = data_D[(kp - start_z) * xdim_weight * ydim_weight + (ip - start_y) * xdim_weight + jp - start_x].y * Fnewweight;

}

void BackProjector::decenter_gpu(CUFFT_COMPLEX * data_D,
                                 CUFFT_COMPLEX * Fconv_D,
                                 double* Fnewweight_D,
                                 int max_r2,
                                 int xdim,
                                 int ydim,
                                 int zdim,
                                 int xdim_weight,
                                 int ydim_weight,
                                 int start_x,
                                 int start_y,
                                 int start_z)
{
	int model_size = xdim * ydim * zdim;
	dim3 blockDim(BLOCK_SIZE_128, 1, 1);
	dim3 gridDim((model_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, 1, 1);
	decenter_kernel <<< gridDim, blockDim>>>(data_D, Fconv_D, Fnewweight_D, max_r2,
	                                         xdim, ydim, zdim, xdim_weight, ydim_weight,
	                                         start_x, start_y, start_z);
}
void BackProjector::convoluteBlobRealSpace_gpu(FourierTransformer& transformer, DOUBLE* Mconv_D, DOUBLE* tabulatedValues_D, bool do_mask)
{

	// Blob normalisation in Fourier space
	DOUBLE normftblob = tab_ftblob(0.);

	transformer.setReal_gpu(Mconv_D, 1, pad_size, pad_size, (ref_dim == 2 ? 1 : pad_size));
	transformer.inverseTransform_gpu();

	Multi_by_FT_tab_gpu(Mconv_D, tabulatedValues_D,  tab_ftblob(0.),  tab_ftblob.sampling, pad_size, pad_size / 2, ori_size * padding_factor, padding_factor, tab_ftblob.tabulatedValues.xdim, false);


	transformer.Transform_gpu(1,  pad_size, pad_size, (ref_dim == 2) ? 1 : pad_size);

}
__global__ void windowFourierTransform_3D_kernel(CUFFT_COMPLEX * in, CUFFT_COMPLEX * out, int ixdim, int iydim, int izdim, int oxdim, int oydim, int ozdim)
{
	int index_within_img = threadIdx.x + blockIdx.x * blockDim.x;
	int i_offset, o_offset;

	i_offset = blockIdx.y * izdim * iydim * ixdim;
	o_offset = blockIdx.y * ozdim * oydim * oxdim;
	if (oxdim > ixdim)
	{
		long int max_r2 = (ixdim - 1) * (ixdim - 1);
		int i, j, k, ip, jp, kp;
		j = index_within_img % ixdim;
		i = (index_within_img / ixdim) % iydim;
		k =  index_within_img / (ixdim * iydim);
		jp = j;
		ip = (i < ixdim) ? i : (i - iydim);
		kp = (k < ixdim) ? k : (k - izdim);

		if (index_within_img >= (ixdim * iydim * izdim) || (kp * kp + ip * ip + jp * jp) > max_r2)
		{
			return;
		}
		int okp = (kp < 0) ? (kp + ozdim) : (kp);
		int oip = (ip < 0) ? (ip + oydim) : (ip);
		int ikp = (kp < 0) ? (kp + izdim) : (kp);
		int iip = (ip < 0) ? (ip + iydim) : (ip);
		out[okp * oydim * oxdim + oip * oxdim + jp + o_offset] =  in[ikp * iydim * ixdim + iip * ixdim + jp + i_offset];

	}
	else
	{
		int i, j, k, ip, jp, kp;
		j = index_within_img % oxdim;
		i = (index_within_img / oxdim) % oydim;
		k =  index_within_img / (oxdim * oydim);
		jp = j;
		ip = (i < oxdim) ? i : (i - oydim);
		kp = (k < oxdim) ? k : (k - ozdim);

		if (index_within_img >= (oxdim * oydim * ozdim))
		{
			return;
		}

		int ikp = (kp < 0) ? (kp + izdim) : (kp);
		int iip = (ip < 0) ? (ip + iydim) : (ip);

		out[index_within_img + o_offset] = in[ikp * iydim * ixdim + iip * ixdim + jp + i_offset];

	}

}
void windowFourierTransform_3D_gpu(CUFFT_COMPLEX * in,
                                   CUFFT_COMPLEX * out,
                                   int newdim,
                                   int nr_images,
                                   int ndim,
                                   int xdim,
                                   int ydim,
                                   int zdim)
{

	if (ydim > 1 && ydim / 2 + 1 != xdim)
	{
		REPORT_ERROR("windowFourierTransform ERROR: the Fourier transform should be of an image with equal sizes in all dimensions!");
	}
	long int newhdim = newdim / 2 + 1;

	if (newhdim == xdim)
	{
		hipMemcpy(out, in, nr_images * zdim * ydim * xdim * sizeof(CUFFT_COMPLEX ), hipMemcpyDeviceToDevice);
		return;
	}
	int out_size = newhdim * newdim * ((ndim == 2) ? 1 : newdim);
	dim3 dimBlock(BLOCK_SIZE_128, 1);
	dim3 dimGrid((out_size + BLOCK_SIZE_128 - 1) / BLOCK_SIZE_128, nr_images, 1);
	windowFourierTransform_3D_kernel <<< dimGrid, dimBlock>>>(in, out,
	                                                          xdim,
	                                                          (ndim > 1 ? ydim : 1),
	                                                          (ndim > 2 ? zdim : 1),
	                                                          newhdim,
	                                                          (ndim > 1 ? newdim : 1),
	                                                          (ndim > 2 ? newdim : 1));

}



void BackProjector::windowToOridimRealSpace_gpu(FourierTransformer& transformer,
                                                CUFFT_COMPLEX * Fin_D, DOUBLE* Mout_D,
                                                int new_xdim,
                                                int new_ydim,
                                                int new_zdim)
{
	int padoridim = padding_factor * ori_size;
	DOUBLE normfft;
	if (ref_dim == 2)
	{
		normfft = (DOUBLE)(padding_factor * padding_factor);
	}
	else
	{
		normfft = (DOUBLE)(padding_factor * padding_factor * padding_factor * ori_size);
	}


	CUFFT_COMPLEX * Ftmp_D;
	int fourier_size = (padoridim / 2 + 1) * padoridim * ((ref_dim == 2) ? 1 : padoridim);
	hipMalloc((void**)&Ftmp_D, fourier_size * sizeof(CUFFT_COMPLEX ));
	hipMemset(Ftmp_D, 0., fourier_size * sizeof(CUFFT_COMPLEX ));
	windowFourierTransform_3D_gpu(Fin_D,
	                              Ftmp_D,
	                              padoridim,
	                              1,
	                              ref_dim,
	                              new_xdim,
	                              new_ydim,
	                              new_zdim); // Do the inverse FFT
	hipfftHandle fPlanBackward_gpu;
#ifdef FLOAT_PRECISION
	hipfftResult fftplan1 = hipfftPlan3d(&fPlanBackward_gpu ,  padoridim, padoridim, (ref_dim == 2 ? 1 : padoridim), HIPFFT_C2R);
	hipfftExecC2R(fPlanBackward_gpu,  Ftmp_D, Mout_D);	
#else
	hipfftResult fftplan1 = hipfftPlan3d(&fPlanBackward_gpu ,  padoridim, padoridim, (ref_dim == 2 ? 1 : padoridim), HIPFFT_Z2D);
	hipfftExecZ2D(fPlanBackward_gpu,  Ftmp_D, Mout_D);
#endif
	
	hipfftDestroy(fPlanBackward_gpu);
	hipFree(Ftmp_D);
	hipError_t cudaStat = hipGetLastError();
	if (cudaStat != hipSuccess)
	{
		printf("kernel calculate_weight_gpu returned error code %d, line(%d), %s\n", cudaStat, __LINE__, hipGetErrorString(cudaStat));
		exit(EXIT_FAILURE);
	}

	DOUBLE* Mout_temp_D;
	if (ref_dim == 2)
	{
		hipMalloc((void**)&Mout_temp_D, padoridim * padoridim * sizeof(DOUBLE)); //Mout.resize(padoridim, padoridim);
	}
	else
	{
		hipMalloc((void**)&Mout_temp_D, padoridim * padoridim * padoridim * sizeof(DOUBLE));
	}

	centerFFT_2_gpu(Mout_D, Mout_temp_D, 1, ref_dim, padoridim, padoridim, ((ref_dim == 2) ? 1 : padoridim), true);

	window_gpu(Mout_temp_D, Mout_D, normfft, FIRST_XMIPP_INDEX(ori_size),  FIRST_XMIPP_INDEX(ori_size),  FIRST_XMIPP_INDEX(ori_size),
	           FIRST_XMIPP_INDEX(padoridim),  FIRST_XMIPP_INDEX(padoridim)
	           , FIRST_XMIPP_INDEX(((ref_dim == 2) ? 1 : padoridim)),
	           ori_size, ori_size, ((ref_dim == 2) ? 1 : ori_size), padoridim, padoridim, ((ref_dim == 2) ? 1 : padoridim));

	softMaskOutsideMap_new_gpu(Mout_D, -1., 3., (DOUBLE*) NULL, 1, ori_size, ori_size, ((ref_dim == 2) ? 1 : ori_size));
	hipFree(Mout_temp_D);
}




